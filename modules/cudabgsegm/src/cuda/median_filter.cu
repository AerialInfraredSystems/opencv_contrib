#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/limits.hpp"

namespace cv
{
namespace cuda
{
namespace device
{
namespace median_filter
{
	
template <class Ptr2D>
__device__ __forceinline__ void swap(Ptr2D &ptr, int x, int y, int a, int b, int rows)
{
    typename Ptr2D::elem_type val = ptr(a * rows + y, x);
    ptr(a * rows + y, x) = ptr(b * rows + y, x);
    ptr(b * rows + y, x) = val;
}

///////////////////////////////////////////////////////////////
// MOG2

__global__ void medianFilter(const PtrStepSzf frame, PtrStepSzf history, int nFrame)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

	history(nFrame * frame.rows + y, x) = frame(y, x);
}

void median_filter_gpu(PtrStepSzf frame, PtrStepSzf history, int nFrame, hipStream_t stream)
{
    dim3 block(32, 8);
    dim3 grid(divUp(frame.cols, block.x), divUp(frame.rows, block.y));
	
	cudaSafeCall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(medianFilter), hipFuncCachePreferL1));
	medianFilter<<<grid, block, 0, stream>>>(frame, history, nFrame);
	
    cudaSafeCall(hipGetLastError());
	
    if (stream == 0)
        cudaSafeCall(hipDeviceSynchronize());
}

__global__ void getMedianImage(PtrStepSzf history, PtrStepSzf dst, int nFrame)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

	int left = 0;
	int right = nFrame - 1;
	int k = nFrame / 2;
   
	while (left <= right) {
		if (left == right) {
            dst(y, x) = history(left * dst.rows + y, x);
            break;
		}

		int pivot = left + (right - left) / 2;
		float pivotValue = history(pivot * dst.rows + y, x);
		swap(history, x, y, pivot, right, dst.rows);
		int storeIndex = left;

		for (int i = left; i < right; i++) {
			if (history(i * dst.rows + y, x) < pivotValue) {
				swap(history, x, y, i, storeIndex, dst.rows);
				storeIndex++;
			}
		}
		swap(history, x, y, storeIndex, right, dst.rows);
		pivot = storeIndex;

		if (pivot == k) {
			dst(y, x) = history(k * dst.rows + y, x);
			break;
		} else if (k < pivot) {
			right = pivot - 1;
		} else {
			left = pivot + 1;
		}	
	}
}

void getMedianImage_gpu(PtrStepSzf history, PtrStepSzf dst, int nFrame, hipStream_t stream)
{
	dim3 block(32, 8);
	dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

    cudaSafeCall(hipFuncSetCacheConfig(reinterpret_cast<const void*>(getMedianImage), hipFuncCachePreferL1));

	getMedianImage<<<grid, block, 0, stream>>>(history, dst, nFrame);

	cudaSafeCall(hipGetLastError());

    if (stream == 0)
        cudaSafeCall(hipDeviceSynchronize());
}
} // namespace median_filter
} // namespace device
} // namespace cuda
} // namespace cv

#endif /* CUDA_DISABLER */
